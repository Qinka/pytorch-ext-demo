#include "hip/hip_runtime.h"
#define _DOT_POWER_C_
#include "dot_power.h"

__global__ static
void kernel_dot_power_forward(const float* base, const float* exponent,
                              float* output,
                              int n) {
  auto index  = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;

  for(auto i = index; i < n; i += stride)
    output[i] = powf(base[i],exponent[i]);
}

__global__ static
void kernel_dot_power_backward(const float* base, const float* exponent,
                               const float* grad_output,
                               float* grad_base, float* grad_exponent,
                               int n) {
  auto index  = blockIdx.x * blockDim.x + threadIdx.x;
  auto stride = blockDim.x * gridDim.x;

  for(auto i = index; i < n; i += stride) {
    grad_base[i] = grad_output[i] * exponent[i] * powf(base[i], exponent[i] - 1);
    grad_exponent[i] = grad_output[i] * logf(exponent[i]) * powf(base[i],exponent[i]);
  }
}

hipError_t dot_power_forward(const float* base, const float* exponent,
                              float* output,
                              int n, hipStream_t stream) {
  kernel_dot_power_forward<<<(n-1)/256+1,256,0,stream>>>(base,exponent,output,n);
  return hipGetLastError();
  //return hipDeviceSynchronize();
}

hipError_t dot_power_backward(const float* base, const float* exponent,
                               const float* grad_output,
                               float* grad_base, float* grad_exponent,
                               int n, hipStream_t stream) {
  kernel_dot_power_backward<<<(n-1)/256+1,256,0,stream>>>(base,exponent,grad_output,
                                                          grad_base,grad_exponent,
                                                          n);
  return hipGetLastError();
  //return hipDeviceSynchronize();
}
